/*
 * an example of persistent cache use case on A100
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <iomanip>

using namespace std;

const int block_num = 1024;
const int block_size = 1024;

#define ENABLE_PERSIST

__global__
void cuda_kernel(float *freq_data, float *stream_data, int freq_size, int stream_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int num = (freq_size + stream_size) / block_size / block_num;
  for(int j = 0; j < num; j++) {
    freq_data[((i * num) + j) % freq_size] = freq_data[((i * num) + j) % freq_size] * 2;
    stream_data[((i * num) + j) % stream_size] = stream_data[((i * num) + j) % stream_size] * 2;
  }
}

int main(int argc, char** argv) {
  int device_id = 0;
  // 128 * 4 MB 
  const int data_size = 1024 * 1024 * 1024 / sizeof(float);
  const int freq_size = 1024 * 1024 * 10 / sizeof(float);
  const int stream_size = data_size - freq_size;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  cudaCtxResetPersistingL2Cache();
  hipStream_t stream;
  hipStreamCreate(&stream);                                                                  // Create CUDA stream

  hipDeviceProp_t prop;                                                                        // CUDA device properties variable
  hipGetDeviceProperties(&prop, device_id);                                                 // Query GPU properties
  cout << "l2 cache size: " << prop.l2CacheSize << endl;
  cout << "max persisting cache size: " << prop.persistingL2CacheMaxSize << " Byte"<< endl;
  cout << "set persisting cache size: " << freq_size * sizeof(float) << " Byte"<< endl;
  hipDeviceSetLimit( cudaLimitPersistingL2CacheSize, freq_size * sizeof(float));                                  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed

  float* h_data = (float *)malloc(data_size * sizeof(float));
  float* data;
  // init host data
  for (int i = 0; i < data_size; i++) {
    h_data[i] = 1.0f;
  }

  hipMalloc(&data, data_size * sizeof(float)); 
  hipMemcpy(data, h_data, data_size * sizeof(float), hipMemcpyHostToDevice);
  
 #ifdef ENABLE_PERSIST 
  hipLaunchAttributeValue stream_attribute;
  stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(data);              // Global Memory data pointer
  stream_attribute.accessPolicyWindow.num_bytes = min((long)(freq_size * sizeof(float)), (long)(prop.accessPolicyMaxWindowSize));                  // Number of bytes for persistence access
  stream_attribute.accessPolicyWindow.hitRatio  = 1.0;                                        // Hint for cache hit ratio
  stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;               // Persistence Property
  stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;                // Type of access property on cache miss
  
  cout << "window num_bytes: " << stream_attribute.accessPolicyWindow.num_bytes << endl;
  cout << "window hit ratio: " << stream_attribute.accessPolicyWindow.hitRatio << endl;
  
  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);   // Set the attributes to a CUDA Stream
#endif
  float accum = 0;
  for(int i = 0; i < 100; i++) {
    hipEventRecord(start);
    
    cuda_kernel <<<block_num, block_size, 0, stream>>> (data, data + freq_size, freq_size, stream_size); // This data1 is used by a kernel multiple times
    hipEventRecord(stop);
    // copy results
    hipMemcpy(h_data, data, data_size * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    accum += milliseconds;
  }
  cout << "Time: " << fixed << setprecision(6) << accum / 100 << " ms" << endl;

  cudaCtxResetPersistingL2Cache();
}


